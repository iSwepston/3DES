#include "hip/hip_runtime.h"
// System includes
#include <stdio.h>
#include <iostream>
#include <assert.h>
#include <unistd.h>
#include <stdint.h>
#include <algorithm>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>

// User includes
#include <definitions.h>

/**
 * @brief doShifting: a device function which is designed to do the "work" of masking a byte
 * @param shiftArray: The shifting directions
 * @param input: input byte array array to shift around
 * @return
 */
__device__ byte doShifting(int * shiftArray, byte * input, int id)
{
    byte output = 0x00;

    // Loop over all bits in current byte
    for(int i = id * 8; i < (id+1)*8; i++) {

        // determine bit to relocate
        int bit = shiftArray[i] - 1;

        // mask and shift
        byte temp = input[bit/8]; // get approprate byte
        temp = temp >> (8-(bit+1)%8)%8; // shift to position 0
        temp &= 0x01; // mask in only this bit

        temp = temp << 7 - i%8; // shift to right position

        // set bits
        output |= temp;
    }

    return output;
}

// Goes from 8 bytes downto 7
__global__ void cuPC1(byte *key, byte *result)
{
    int id = get_idx();

    result[id] = doShifting(PC_1, key, id);
}

// one thread per key
__global__ void cuLeftCircShifts(byte * input, byte * result)
{
    int id = get_idx();
    int shamt = id + 1;

    uint upper = 0;
    uint lower = 0;

    for(int i = 0; i <= 3; i++) {
        uint temp = 0;
        temp += input[i];

        temp = temp << (3-i)*8;
        upper |= temp;
    }

    upper = upper >> 4;

    for(int i = 3; i <= 6; i++) {
        uint temp = 0;
        temp |= input[i];
        temp = temp << (6 - i)*8;
        lower |= temp;
    }

    // mask out
    lower &= 0x0FFFFFFF;

    int numbits = 28; //7 nibbles
    uint mask = upper >> numbits - shamt;
    upper = upper << shamt;
    upper |= mask;
    upper &= 0x0FFFFFFF;

    mask = lower >> numbits - shamt;
    lower = lower << shamt;
    lower |= mask;
    lower &= 0x0FFFFFFF;

    uint64_t out = upper;
    out <<= numbits;
    out |= lower;

    int start = id*7;
    for(int i = start; i < start+7; i++) {
        result[i] = (out >> (6 - i%7)*8) & 0x0FF;
    }

    //    printf("id %d: %lx\n", id, out);
}

__global__ void cuPC2(byte * input, byte * round_keys)
{
    int id = get_idx();
    int round = (id/6);

    //    byte * x = &input[round*7];
    //    for(int i = 0; i < 7; i++) {
    //        if(id==7) printf("%02X\n", x[i]);
    //    }

    byte temp = doShifting(PC_2, &input[round*7], id%6);
    //    if(id == 7) printf("Res: %0x\n", temp);
    round_keys[id] = temp;
}

// 64 -> 64
__global__ void cuIP(byte * input, byte * output)
{
    int id = get_idx();

    output[id] = doShifting(IP, input, id);

    printf("IP: id = %d: %02x\n", id, output[id]);
}

// 32(64) -> 48
__global__ void cuEPerm(byte * input, byte * output)
{
    int id = get_idx();

    output[id] = doShifting(Ex, input, id%3);

    printf("Ex: id = %d: %02x\n", id, output[id]);
}

// 48 -> 48
__global__ void cuXOR(byte * input, byte * key, byte * output)
{
    int id = get_idx();

    output[id] = input[id] ^ key[id%6];

    printf("XOR: id = %d: %02x\n", id, output[id]);
}

// 48 -> 32
__global__ void cuSBoxes(byte * input, byte * output)
{
    int id = get_idx();
    output[id] = 0;

    uint64_t value = 0;
    for(int i = 0; i < 6; i++) {
        value |= input[i];
        value = value << 8;
    }

    value = value >> 8;

    int modId = id % 8;
    byte test = (value >> (7 - modId)*6) & 0x3F;
    byte row = ((test >> 4) & 0x02) | (test & 0x01); // outer bits
    byte col = (test >> 1) & 0x0F; // inner bits

//    if (id == 0) printf("%lx\n", value);

    int offset = row * 16 + col;
    byte result = boxes[id%8][offset];

    output[id] |= result << 4*(1 - (id % 2));
    __syncthreads();

    if(id % 2 == 0)
        output[id] = output[id] | output[id + 1];

    __syncthreads();

    if(id % 8 == 0) {
        output[id + 1] = output[id + 2];
        output[id + 2] = output[id + 4];
        output[id + 3] = output[id + 6];

        printf("S-BOX: %02X%02X%02X%02X\n", output[id + 0], output[id + 1], output[id + 2], output[id + 3]);
    }

//    printf("SBOX: id = %d: %02X -> shamt %d row: %d, col %d, res: %x, out %02x\n", id, test, (7 - modId)*6, row, col, result, output[id]);
}

__global__ void cuPPerm(byte * input, byte * output)
{
    int id = get_idx();

    output[id] = doShifting(Pf, input, id%4);

    printf("P: id = %d: %02x\n", id, output[id]);
}

__global__ void cuCombine(byte * L, byte * R, byte * output)
{
    int id = get_idx();

    if(id % 8 < 4) output[id] = L[id%4];
    else output[id] = R[id%4];

    printf("output: id = %d: %02x\n", id, output[id]);
}

int main(int argc, char **argv)
{

    std::cout << "Testing" << std::endl;

    byte * inputKey;
    byte * output;
    byte * after_shift;
    byte * round_keys;
    hipMalloc((void**)&inputKey, sizeof(byte)*8);
    hipMalloc((void**)&output, sizeof(byte)*7);
    hipMalloc((void**)&after_shift, sizeof(byte)*16*7);
    hipMalloc((void**)&round_keys, sizeof(byte)*16*6);

    byte * holding1;
    byte * holding2;
    byte * holding3;
    hipMalloc((void**)&holding1, sizeof(byte)*8);
    hipMalloc((void**)&holding2, sizeof(byte)*8);
    hipMalloc((void**)&holding3, sizeof(byte)*8);


    byte key[8] = {0xFE, 0xDC, 0xBA, 0x98, 0x76, 0x54, 0x32, 0x10};

    byte plaintext[] = {0xFE, 0xDC, 0xBA, 0x98, 0x76, 0x54, 0x32, 0x10};

    hipMemcpy(inputKey, &key, sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(holding1, &plaintext, sizeof(uint64_t), hipMemcpyHostToDevice);

    // Calculate all keys
    cuPC1<<<1,7>>>(inputKey, output);
    cuLeftCircShifts<<<1,16>>>(output, after_shift);
    cuPC2<<<1,96>>>(after_shift, round_keys); // 96 = 16*6

    cuIP<<<1,8>>>(holding1, holding2);
    byte * L0 = holding2;
    byte * R0 = &holding2[4];

    cuEPerm<<<1,6>>>(R0, holding3);
    cuXOR<<<1,6>>>(holding3, round_keys, holding1);

    cuSBoxes<<<1,8>>>(holding1, holding3);
    cuPPerm<<<1,4>>>(holding3, holding1);

    cuXOR<<<1,4>>>(holding1, L0, holding3);
    cuCombine<<<1, 8>>>(R0, holding3, holding1);



    hipDeviceSynchronize();

    byte result[16*6];
    hipMemcpy(result, round_keys, 16*6, hipMemcpyDeviceToHost);

    //    printf("Output: ");
    //    for(int i = 0; i < 16; i++) {
    //        printf("Key %d:\n", i+1);

    //        for(int j = i*6; j < i*6 + 6; j++)
    //            printf("%02X",result[j]);

    //        printf("\n");
    //    }

    //    printf("\n");

    return 0;
}
